#include "define.h"
#include "DataProcessing.h"
#include "Jacob.h"
#include "measure.h";
#include "LUdec.h"
#include "appendix.h"


int main() {
	LARGE_INTEGER freq, start, stop;
	double exe_time;
	QueryPerformanceFrequency(&freq);
	QueryPerformanceCounter(&start);

	/*Step 1: read the data, including: mat B, mat G, vector P, vector Q, vector vol, vector theta*/
	printf("***The accelaration of State Estimation based on GPU***\n");
	printf("			writer:linm\n\n");
	printf("PreSE start\n	Reading and processing data...\n");
	struct point *data;
	FILE *mat;
	double SEtime;
	mat = fopen("case/case13659.txt", "r");
	if (mat == NULL) {
		printf("fail");
	}
	int nB, numOfData;
	fscanf(mat, "%d", &nB);
	fscanf(mat, "%d", &numOfData);
	char s[100];
	fgets(s, 100, mat);
	data = (point*)malloc(numOfData * sizeof(point));
	for (int i = 0; i < numOfData; i++) {
		fscanf(mat, "%d", &(data[i].rowInd));
		fscanf(mat, "%d", &(data[i].colInd));
		fscanf(mat, "%lf", &(data[i].G));
		fscanf(mat, "%lf", &(data[i].Br));
		fscanf(mat, "%lf", &(data[i].Ba));
		fscanf(mat, "%lf", &(data[i].P));
		fscanf(mat, "%lf", &(data[i].Q));
		fscanf(mat, "%lf", &(data[i].yc_half));
		fscanf(mat, "%lf", &(data[i].vol));
		fscanf(mat, "%lf", &(data[i].theta));
	}
	//get admittance mat B, power flow vector P,Q, yc
	int *idx2row, *idx2idx;
	my_type *valB, *valG, *P, *Q, *yc_half, *valBa;
	int *colIndB, *rowPtrB, maxPerRowB;
	dataProcessing(data, &idx2row, &idx2idx, &valB, &valG, &valBa, &P, &Q, &yc_half,
		&rowPtrB, &colIndB, nB, numOfData, &maxPerRowB);

	//set the initial value
	my_type *vol, *theta;
	vol = (my_type*)malloc(nB * sizeof(my_type));
	theta = (my_type*)calloc(nB, sizeof(my_type));
	for (int i = 0; i < nB; i++) {
		vol[i] = data[numOfData - 1].vol;
		theta[i] = 0;
	}

	//synchronize the gpu and cpu
	QueryPerformanceCounter(&stop);
	exe_time = 1e3*(stop.QuadPart - start.QuadPart) / freq.QuadPart;
	printf("	Data ready. Spent %f ms \n	Starting the GPU...\n",exe_time);
	SEtime = exe_time;
	QueryPerformanceCounter(&start);

	hipDeviceSynchronize();
	hipsparseHandle_t handle;
	hipsparseCreate(&handle);

	QueryPerformanceCounter(&stop);
	exe_time = 1e3*(stop.QuadPart - start.QuadPart) / freq.QuadPart;
	SEtime += exe_time;
	printf("	GPU ready. Spent %f ms\nPreSE done. Spent %f ms\nSE start\n	Initializing...\n",exe_time,SEtime);
	QueryPerformanceCounter(&start);

	//Get the jacob mat H
	my_type *valH, *valHQ;
	int *rowPtrH, *colIndH;

	GetJacob(valBa, valB, colIndB, rowPtrB, &valH, &valHQ, &rowPtrH, &colIndH, nB);


	//LU refact 
	my_type *valR, *valL, *valRQ, *valLQ;
	int *colIndR, *rowPtrR, *colIndL, *rowPtrL;
	int nH = nB - 1;
	int m = rowPtrB[nB];
	int *Pinv;
	LUdecom(valH, valHQ, rowPtrH, colIndH, &Pinv, &valL, &colIndL, &rowPtrL, &valR, &colIndR, &rowPtrR, &valLQ, &valRQ, m, nH);

	//reorder the jacob mat H

	reorder(valH, valHQ, rowPtrH, colIndH, Pinv, m, nH);

	

	/*step3: loop*/
	hipsparseStatus_t status;
	/*copy L U mat*/
	int nnzR = rowPtrR[nH];
	int nnzL = rowPtrL[nH];
	my_type *d_valR, *d_valRQ, *d_valL, *d_valLQ;
	int *d_colIndR, *d_rowPtrR, *d_colIndL, *d_rowPtrL;
	hipMalloc((void**)&d_valR, nnzR * sizeof(my_type));
	hipMalloc((void**)&d_valRQ, nnzR * sizeof(my_type));
	hipMalloc((void**)&d_colIndR, nnzR * sizeof(int));
	hipMalloc((void**)&d_rowPtrR, (nH + 1) * sizeof(int));
	hipMemcpy(d_valR, valR, nnzR * sizeof(my_type), hipMemcpyHostToDevice);
	hipMemcpy(d_valRQ, valRQ, nnzR * sizeof(my_type), hipMemcpyHostToDevice);
	hipMemcpy(d_colIndR, colIndR, nnzR * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_rowPtrR, rowPtrR, (nH + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_valL, nnzL * sizeof(my_type));
	hipMalloc((void**)&d_valLQ, nnzL * sizeof(my_type));
	hipMalloc((void**)&d_colIndL, nnzL * sizeof(int));
	hipMalloc((void**)&d_rowPtrL, (nH + 1) * sizeof(int));
	hipMemcpy(d_valL, valL, nnzL * sizeof(my_type), hipMemcpyHostToDevice);
	hipMemcpy(d_valLQ, valLQ, nnzL * sizeof(my_type), hipMemcpyHostToDevice);
	hipMemcpy(d_colIndL, colIndL, nnzL * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_rowPtrL, rowPtrL, (nH + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipsparseMatDescr_t descrR;
	hipsparseCreateMatDescr(&descrR);
	hipsparseSetMatType(descrR, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatFillMode(descrR, HIPSPARSE_FILL_MODE_UPPER);
	hipsparseSetMatIndexBase(descrR, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatDiagType(descrR, HIPSPARSE_DIAG_TYPE_UNIT);
	hipsparseMatDescr_t descrL;
	hipsparseCreateMatDescr(&descrL);
	hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_NON_UNIT);

	/*copy H mat*/
	int nnzH = rowPtrH[m];
	my_type *d_valH, *d_valHQ;
	int *d_colIndH, *d_rowPtrH;
	hipMalloc((void**)&d_valH, nnzH * sizeof(my_type));
	hipMalloc((void**)&d_valHQ, nnzH * sizeof(my_type));
	hipMalloc((void**)&d_colIndH, nnzH * sizeof(int));
	hipMalloc((void**)&d_rowPtrH, (m + 1) * sizeof(int));
	hipMemcpy(d_valH, valH, nnzH * sizeof(my_type), hipMemcpyHostToDevice);
	hipMemcpy(d_valHQ, valHQ, nnzH * sizeof(my_type), hipMemcpyHostToDevice);
	hipMemcpy(d_colIndH, colIndH, nnzH * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_rowPtrH, rowPtrH, (m + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipsparseMatDescr_t descrH;
	hipsparseCreateMatDescr(&descrH);

	/*copy B mat*/
	int nnzB = rowPtrB[nB];
	my_type *d_valB, *d_valG;
	int *d_colIndB, *d_rowPtrB;
	hipMalloc((void**)&d_valG, nnzB * sizeof(my_type));
	hipMalloc((void**)&d_valB, nnzB * sizeof(my_type));
	hipMalloc((void**)&d_rowPtrB, (nB + 1) * sizeof(int));
	hipMalloc((void**)&d_colIndB, nnzB * sizeof(int));
	hipMemcpy(d_valG, valG, nnzB * sizeof(my_type), hipMemcpyHostToDevice);
	hipMemcpy(d_valB, valB, nnzB * sizeof(my_type), hipMemcpyHostToDevice);
	hipMemcpy(d_rowPtrB, rowPtrB, (nB + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_colIndB, colIndB, nnzB * sizeof(int), hipMemcpyHostToDevice);

	/*copy P,Q,vol,theta,yc*/
	my_type *d_P, *d_Q, *d_vol, *d_theta, *d_yc_half;
	hipMalloc((void**)&d_P, m * sizeof(my_type));
	hipMalloc((void**)&d_Q, m * sizeof(my_type));
	hipMemcpy(d_P, P, m * sizeof(my_type), hipMemcpyHostToDevice);
	hipMemcpy(d_Q, Q, m * sizeof(my_type), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_vol, nB * sizeof(my_type));
	hipMalloc((void**)&d_theta, nB * sizeof(my_type));
	hipMemcpy(d_vol, vol, nB * sizeof(my_type), hipMemcpyHostToDevice);
	hipMemcpy(d_theta, theta, nB * sizeof(my_type), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_yc_half, m * sizeof(my_type));
	hipMemcpy(d_yc_half, yc_half, m * sizeof(my_type), hipMemcpyHostToDevice);

	/*prepare for calculating measurement function*/
	my_type *d_caledP, *d_caledQ;
	int *d_idx2row, *d_idx2idx;
	hipMalloc((void**)&d_idx2row, m * sizeof(int));
	hipMemcpy(d_idx2row, idx2row, m * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_idx2idx, m * sizeof(int));
	hipMemcpy(d_idx2idx, idx2idx, m * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_caledP, m * sizeof(my_type));
	hipMalloc((void**)&d_caledQ, m * sizeof(my_type));


	/*prepare for calculating difference vector*/
	my_type *d_dP, *d_dQ, *d_J;
	hipMalloc((void**)&d_J, sizeof(my_type));
	hipMalloc((void**)&d_dP, m * sizeof(my_type));
	hipMalloc((void**)&d_dQ, m * sizeof(my_type));


	/*prepare for calculating the b vector*/
	my_type alpha = 1, beta = 0;
	my_type *d_bP, *d_bQ;
	hipMalloc((void**)&d_bP, nH * sizeof(my_type));
	hipMalloc((void**)&d_bQ, nH * sizeof(my_type));

	/*prepare for solving LU equation*/
	/*buffersize analysis*/
	int buffersizeL, buffersizeU, buffersizeLQ, buffersizeUQ;
	csrsv2Info_t infoL, infoU, infoLQ, infoUQ;
	hipsparseCreateCsrsv2Info(&infoL);
	hipsparseCreateCsrsv2Info(&infoU);
	hipsparseCreateCsrsv2Info(&infoLQ);
	hipsparseCreateCsrsv2Info(&infoUQ);
	status = hipsparseDcsrsv2_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, nH, nnzL, descrL, d_valL, d_rowPtrL, d_colIndL, infoL, &buffersizeL);
	status = hipsparseDcsrsv2_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, nH, nnzR, descrR, d_valR, d_rowPtrR, d_colIndR, infoU, &buffersizeU);
	status = hipsparseDcsrsv2_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, nH, nnzL, descrL, d_valLQ, d_rowPtrL, d_colIndL, infoLQ, &buffersizeLQ);
	status = hipsparseDcsrsv2_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, nH, nnzR, descrR, d_valRQ, d_rowPtrR, d_colIndR, infoUQ, &buffersizeUQ);
	void * buffer, *bufferQ;
	if (buffersizeL > buffersizeU)
		hipMalloc((void**)&buffer, buffersizeL);
	else
		hipMalloc((void**)&buffer, buffersizeU);
	if (buffersizeLQ > buffersizeUQ)
		hipMalloc((void**)&bufferQ, buffersizeLQ);
	else
		hipMalloc((void**)&bufferQ, buffersizeUQ);
	//the delta vector
	my_type *d_temp, *d_dvol, *d_dtheta;
	hipMalloc((void**)&d_temp, nH * sizeof(my_type));
	hipMalloc((void**)&d_dvol, nH * sizeof(my_type));
	hipMalloc((void**)&d_dtheta, nH * sizeof(my_type));
	/*solving analysis*/
	status = hipsparseDcsrsv2_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, nH, nnzL, descrL, d_valL, d_rowPtrL, d_colIndL, infoL, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, buffer);
	status = hipsparseDcsrsv2_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, nH, nnzR, descrR, d_valR, d_rowPtrR, d_colIndR, infoU, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, buffer);
	status = hipsparseDcsrsv2_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, nH, nnzL, descrL, d_valLQ, d_rowPtrL, d_colIndL, infoLQ, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, bufferQ);
	status = hipsparseDcsrsv2_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, nH, nnzR, descrR, d_valRQ, d_rowPtrR, d_colIndR, infoUQ, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, bufferQ);

	/*prepare for checking the result*/
	int *d_Pinv;
	hipMalloc((void**)&d_Pinv, nB * sizeof(int));
	hipMemcpy(d_Pinv, Pinv, nB * sizeof(int), hipMemcpyHostToDevice);
	//flag for loop
	bool *flagV, *d_flagV, *flagT, *d_flagT;
	flagV = (bool*)malloc(sizeof(bool));
	*flagV = true;
	hipMalloc((void**)&d_flagV, sizeof(bool));
	flagT = (bool*)malloc(sizeof(bool));
	*flagT = true;
	hipMalloc((void**)&d_flagT, sizeof(bool));

	QueryPerformanceCounter(&stop);
	exe_time = 1e3*(stop.QuadPart - start.QuadPart) / freq.QuadPart;
	SEtime = exe_time;
	printf("	Initialization done. Spent %f ms\n	Looping...\n", exe_time);
	QueryPerformanceCounter(&start);

	/*loop*/
	int loop = 0;
	while (*flagV || *flagT) {
		if (*flagT) {
			measureP << <BLOCK, THREAD >> >(nB, d_valG, d_valB, d_rowPtrB, d_colIndB, d_vol, d_theta,
				d_caledP, d_idx2row, d_idx2idx);
			differ << <BLOCK, THREAD >> >(m, d_P, d_caledP, d_dP);
			hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_TRANSPOSE, m, nH, nnzH, &alpha, descrH, d_valH,
				d_rowPtrH, d_colIndH, d_dP, &beta, d_bP);
			status = hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, nH, nnzL, &alpha, descrL,
				d_valL, d_rowPtrL, d_colIndL, infoL, d_bP, d_temp, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, buffer);
			status = hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, nH, nnzR, &alpha, descrR, 
				d_valR, d_rowPtrR, d_colIndR, infoU, d_temp, d_dtheta, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, buffer);
			check << <1, 1024 >> >(d_dtheta, d_theta, nH, d_flagT, d_Pinv);
			hipMemcpy(flagT, d_flagT, sizeof(bool), hipMemcpyDeviceToHost);

		}
		if (*flagV) {
			measureQ << <BLOCK, THREAD >> >(nB, d_valG, d_valB, d_rowPtrB, d_colIndB, d_vol, 
				d_theta, d_caledQ, d_yc_half, d_idx2row, d_idx2idx);
			differ << <BLOCK, THREAD >> >(m, d_Q, d_caledQ, d_dQ);
			hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_TRANSPOSE, m, nH, nnzH, &alpha, descrH, 
				d_valHQ, d_rowPtrH, d_colIndH, d_dQ, &beta, d_bQ);
			status = hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, nH, nnzL, &alpha, descrL,
				d_valLQ, d_rowPtrL, d_colIndL, infoLQ, d_bQ, d_temp, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, bufferQ);
			status = hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, nH, nnzR, &alpha, descrR,
				d_valRQ, d_rowPtrR, d_colIndR, infoUQ, d_temp, d_dvol, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, bufferQ);
			check <<<1, 1024 >>>(d_dvol, d_vol, nH, d_flagV, d_Pinv);
			hipMemcpy(flagV, d_flagV, sizeof(bool), hipMemcpyDeviceToHost);
		}
		loop++;
	}

	QueryPerformanceCounter(&stop);
	exe_time = 1e3*(stop.QuadPart - start.QuadPart) / freq.QuadPart;
	printf("	%d loops done. Spent %f ms\n",loop, exe_time);
	SEtime += exe_time;
	printf("SE done. Spent %f ms\n", SEtime);
	hipMemcpy(vol, d_vol, nB * sizeof(my_type), hipMemcpyDeviceToHost);
	hipMemcpy(theta, d_theta, nB * sizeof(my_type), hipMemcpyDeviceToHost);
	hipMemcpy(P, d_caledP, m * sizeof(my_type), hipMemcpyDeviceToHost);
	hipMemcpy(Q, d_caledQ, m * sizeof(my_type), hipMemcpyDeviceToHost);

	/*clear*/
	hipFree(d_valL);
	hipFree(d_valLQ);
	hipFree(d_colIndL);
	hipFree(d_rowPtrL);
	hipFree(d_valR);
	hipFree(d_valRQ);
	hipFree(d_colIndR);
	hipFree(d_rowPtrR);
	hipFree(d_valB);
	hipFree(d_colIndB);
	hipFree(d_rowPtrB);
	hipFree(d_valH);
	hipFree(d_valHQ);
	hipFree(d_colIndH);
	hipFree(d_rowPtrH);
	hipFree(d_valG);
	hipFree(d_P);
	hipFree(d_Q);
	hipFree(d_vol);
	hipFree(d_theta);
	hipFree(d_caledP);
	hipFree(d_caledQ);
	hipFree(d_idx2row);
	hipFree(d_idx2idx);
	hipFree(d_J);
	hipFree(d_dP);
	hipFree(d_dQ);
	hipFree(d_bP);
	hipFree(d_bQ);
	hipFree(d_temp);
	hipFree(d_dvol);
	hipFree(d_dtheta);
	hipFree(d_yc_half);
	hipFree(d_Pinv);
	hipFree(d_flagT);
	hipFree(d_flagV);
}